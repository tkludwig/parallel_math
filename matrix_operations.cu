#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>
#include <cmath>

#include "matrix_operations.cuh"

int random_uniform_initialize(double* m, int M, int N, double lower, double upper)
{
	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(lower, upper);

	for (int i = 0; i < M * N; ++i)
	{
		*(m + i) = distribution(generator);
	}
	return 0;
}

int naive_multiply_sequential(double* A, double* B, double* C, int M, int K, int N)
{
	for (int i = 0; i < M * N; ++i)
	{
		double val = 0.;
		int row = i % M;
		int col = i / M;
		for (int j = 0; j < K; ++j)
		{
			val += *(A + j * M + row) * *(B + col * K + j);
		}
		*(C + i) = val;
	}
	return 0;
}

int naive_multiply_omp(double* A, double* B, double* C, int M, int K, int N)
{
#pragma omp parallel for
	for (int i = 0; i < M * N; ++i)
	{
		//printf("hello from thread %d", omp_get_thread_num());
		double val = 0.;
		int row = i % M;
		int col = i / M;
		for (int j = 0; j < K; ++j)
		{
			val += *(A + j * M + row) * *(B + col * K + j);
		}
		*(C + i) = val;
	}
	return 0;
}

void print_matrix(const double* mat, int M, int N)
{
	std::cout << std::endl;
	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			std::cout << std::fixed << std::setprecision(3) << std::setw(8) << *(mat + j * M + i) << " ";
		}
		std::cout << std::endl;
	}
}

void print_matrix_big(const double* mat, int M, int N, int include)
{
	std::cout << std::endl << "[" << std::endl;
	for (int i = 0; i < include; ++i)
	{
		for (int j = 0; j < include; ++j)
		{
			std::cout << std::fixed << std::setprecision(3) << std::setw(8) << *(mat + j * M + i) << " ";
		}
		std::cout << "        ..." << std::endl;
	}
	std::cout << "                ...";
	for (int i = (M - include); i < M; ++i)
	{
		std::cout << std::endl << "                ...    ";
		for (int j = (N - include); j < N; ++j)
		{
			std::cout << std::fixed << std::setprecision(3) << std::setw(8) << *(mat + j * M + i) << " ";
		}
	}
	std::cout << std::endl << "]" << std::endl;
}

double max_diff(const double* mat1, const double* mat2, int M, int N)
{
	double maxd = 0.;
	double tdiff = 0.;
	for (int i = 0; i < M * N; ++i)
	{
		tdiff = std::abs(*(mat1 + i) - *(mat2 + i));
		if (tdiff > maxd)
		{
			maxd = tdiff;
		}
	}
	return maxd;
}