/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>

#include "matrix_operations.cuh"

int main()
{
	int M = 8;
	int N = 8;
	double mat1[64];
	int init_failed = random_uniform_initialize(mat1, M, N, 0.0, 10.5);
	print_matrix(mat1, 8, 8);
	print_matrix_big(mat1, 8, 8, 3);

	std::cout << "test sequential naive multiply:" << std::endl;
	double A[4] = { 2., 0., 0., 3. };
	double B[4] = { 2., 0., 1., 2. };
	double C[4];
	naive_multiply_sequential(A, B, C, 2, 2, 2);
	print_matrix(C, 2, 2);
}