/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>