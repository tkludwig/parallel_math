/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>

int random_uniform_initialize(double* m, int M, int N, double lower, double upper)
{
	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(lower, upper);

	for (int i = 0; i < M * N; ++i)
	{
		*(m + i) = distribution(generator);
	}
	return 0;
}

int sequential_naive_multiply(double* A, double* B, double* C, int M, int K, int N)
{
	for (int i = 0; i < M * N; ++i)
	{
		double val = 0.;
		int row = i % M;
		int col = i / M;
		for (int j = 0; j < K; ++j)
		{
			val += *(A + j * M + row) * *(B + col * K + j);
		}
		*(C + i) = val;
	}
	return 0;
}

int main()
{
	int M = 50;
	int N = 60;
	double mat1[3000];
	std::cout << "initial, final values of mat1: " << mat1[0] << " " << mat1[1] << " " <<
		mat1[2] << " ... " << mat1[2997] << " " << mat1[2998] << " " << mat1[2999] << std::endl;
	int init_failed = random_uniform_initialize(mat1, M, N, 0.0, 0.5);
	std::cout << "initial, final values of mat1: " << mat1[0] << " " << mat1[1] << " " <<
		mat1[2] << " ... " << mat1[2997] << " " << mat1[2998] << " " << mat1[2999] << std::endl;

	std::cout << "test sequential naive multiply:" << std::endl;
	double A[4] = { 2., 0., 0., 3. };
	double B[4] = { 2., 0., 1., 2. };
	double C[4];
	sequential_naive_multiply(A, B, C, 2, 2, 2);
	std::cout << "C: " << C[0] << " " << C[1] << " " << C[2] << " " << C[3] << std::endl;
}