/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>

#include "matrix_operations.cuh"

int main()
{
	const int M = 99;
	const int N = 99;
	const int K = 99;
	double mat1[M * K];
	double mat2[K * N];
	double mat3[M * N]; //array to store result from sequential multiply
	double mat4[M * N]; //array to store result from omp multiply
	double mat5[M * N]; //host array to store result from gpu multiply

	double* d_mat1; //device pointer
	double* d_mat2;
	double* d_mat3;

	hipMalloc(&d_mat1, M * N * sizeof(double));
	hipMalloc(&d_mat2, M * N * sizeof(double));
	hipMalloc(&d_mat3, M * N * sizeof(double));

	random_uniform_initialize(mat1, M, N, 0.0, 0.5);
	random_uniform_initialize(mat2, M, N, 0.0, 0.5);

	std::cout << "mat1 initial:" << std::endl;
	print_matrix_big(mat1, M, N, 3);

	std::cout << "test sequential naive multiply:" << std::endl;
	auto start = std::chrono::steady_clock::now();
	naive_multiply_sequential(mat1, mat2, mat3, M, K, N);
	auto end = std::chrono::steady_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_sequential on size " << M << " took " << duration.count() << " ms" << std::endl;

	std::cout << "test omp naive multiply:" << std::endl;
	start = std::chrono::steady_clock::now();
	naive_multiply_omp(mat1, mat2, mat4, M, K, N);
	end = std::chrono::steady_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_omp on size " << M << " took " << duration.count() << " ms" << std::endl;

	std::cout << "test cuda naive multiply:" << std::endl;
	//ordinarily I would wrap the hipMemcpy into the host function, but I want to see actually how long these mem copies take
	auto start_kernel = std::chrono::steady_clock::now();
	auto end_kernel = std::chrono::steady_clock::now();

	start = std::chrono::steady_clock::now();


	hipMemcpy(d_mat1, mat1, M * K * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_mat2, mat2, K * N * sizeof(double), hipMemcpyHostToDevice);

	start_kernel = std::chrono::steady_clock::now();
	naive_multiply_cuda(d_mat1, d_mat2, d_mat3, M, K, N);
	end_kernel = std::chrono::steady_clock::now();
	hipMemcpy(mat5, d_mat3, M * N * sizeof(double), hipMemcpyDeviceToHost);

	end = std::chrono::steady_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	auto kernel_duration = std::chrono::duration_cast<std::chrono::microseconds>(end_kernel - start_kernel);

	std::cout << "naive_multiple_cuda on size " << M << " took " << duration.count() << " ms" << std::endl;
	std::cout << "naive_multiple_cuda kernel on size " << M << " took " << kernel_duration.count() << " ms" << std::endl;

	print_matrix_big(mat3, M, N, 3);
	print_matrix_big(mat4, M, N, 3);
	print_matrix_big(mat5, M, N, 3);
}