/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>
#include <armadillo>

#include "matrix_operations.cuh"

int main()
{
	std::cout << "Armadillo version: " << arma::arma_version::as_string() << std::endl;
	const int M = 79;
	const int N = 79;
	double mat1[M*N];
	double mat2[M*N];
	double mat3[M*N];
	int init_failed = random_uniform_initialize(mat1, M, N, 0.0, 0.5);

	std::cout << "test sequential naive multiply:" << std::endl;
	double A[4] = { 2., 0., 0., 3. };
	double B[4] = { 2., 0., 1., 2. };
	double C[4];
	naive_multiply_sequential(A, B, C, 2, 2, 2);
	print_matrix(C, 2, 2);

	std::cout << "test omp naive multiply:" << std::endl;
	auto start = std::chrono::high_resolution_clock::now();
	naive_multiply_omp(mat1, mat1, mat2, M, N, N);
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_omp on size " << M << " took " << duration.count() << " ms" << std::endl;

	start = std::chrono::high_resolution_clock::now();
	naive_multiply_sequential(mat1, mat1, mat3, M, N, N);
	end = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_sequential on size " << M << " took " << duration.count() << " ms" << std::endl;

	double diff = max_diff(mat2, mat3, M, N);
	std::cout << "diff between sequential and omp matrices: " << diff << std::endl;

	print_matrix_big(mat2, M, N, 3);
	print_matrix_big(mat3, M, N, 3);
}