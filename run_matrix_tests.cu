/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>

int random_uniform_initialize(double* m, int M, int N, double lower, double upper)
{
	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(lower, upper);

	for (int i = 0; i < M * N; ++i)
	{
		*(m + i) = distribution(generator);
	}
	return 0;
}

int main()
{
	int M = 50;
	int N = 60;
	double mat1[3000];
	std::cout << "initial, final values of mat1: " << mat1[0] << " " << mat1[1] << " " <<
		mat1[2] << " ... " << mat1[2997] << " " << mat1[2998] << " " << mat1[2999] << std::endl;
	int init_failed = random_uniform_initialize(mat1, M, N, 0.0, 0.5);
	std::cout << "initial, final values of mat1: " << mat1[0] << " " << mat1[1] << " " <<
		mat1[2] << " ... " << mat1[2997] << " " << mat1[2998] << " " << mat1[2999] << std::endl;
}