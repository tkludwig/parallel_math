/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>


#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>

int random_uniform_initialize(double* m, int M, int N, double lower, double upper)
{
	std::default_random_engine generator;
	std::uniform_real_distribution<double> distribution(lower, upper);

	for (int i = 0; i < M * N; ++i)
	{
		*(m + i) = distribution(generator);
	}
	return 0;
}

int sequential_naive_multiply(double* A, double* B, double* C, int M, int K, int N)
{
	for (int i = 0; i < M * N; ++i)
	{
		double val = 0.;
		int row = i % M;
		int col = i / M;
		for (int j = 0; j < K; ++j)
		{
			val += *(A + j * M + row) * *(B + col * K + j);
		}
		*(C + i) = val;
	}
	return 0;
}

void print_matrix(const double* mat, int M, int N)
{
	//std::cout << std::setw(8);
	std::cout << std::endl;
	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			std::cout << std::fixed << std::setprecision(3) << std::setw(8) << *(mat + j * M + i) << " ";
		}
		std::cout << std::endl;
	}
}

int main()
{
	int M = 8;
	int N = 8;
	double mat1[64];
	int init_failed = random_uniform_initialize(mat1, M, N, 0.0, 10.5);
	print_matrix(mat1, 8, 8);


	std::cout << "test sequential naive multiply:" << std::endl;
	double A[4] = { 2., 0., 0., 3. };
	double B[4] = { 2., 0., 1., 2. };
	double C[4];
	sequential_naive_multiply(A, B, C, 2, 2, 2);
	print_matrix(C, 2, 2);
}