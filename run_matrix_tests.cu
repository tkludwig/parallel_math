/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>
#include <armadillo>

#include "matrix_operations.cuh"

int main()
{

	std::cout << "Armadillo version??: " << arma::arma_version::as_string() << std::endl;
	const int M = 9;
	const int N = 9;
	double mat1[M * N];
	double mat2[M * N];
	double mat3[M * N];
	double mat4[M * N];
	random_uniform_initialize(mat1, M, N, 0.0, 0.5);
	random_uniform_initialize(mat2, M, N, 0.0, 0.5);

	arma::Mat<double> a_mat1(mat1, M, N, true);
	arma::Mat<double> a_mat2(mat2, M, N, true);
	arma::Mat<double> a_mat3(M, N);

	std::cout << "mat1 initial:" << std::endl;
	print_matrix_big(mat1, M, N, 3);
	std::cout << "a_mat1 initial:" << std::endl;
	print_matrix_big(a_mat1.memptr(), M, N, 3);

	std::cout << "test omp naive multiply:" << std::endl;
	auto start = std::chrono::high_resolution_clock::now();
	naive_multiply_omp(mat1, mat2, mat3, M, N, N);
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_omp on size " << M << " took " << duration.count() << " ms" << std::endl;

	start = std::chrono::high_resolution_clock::now();
	naive_multiply_sequential(mat1, mat2, mat4, M, N, N);
	end = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_sequential on size " << M << " took " << duration.count() << " ms" << std::endl;

	a_mat3 = a_mat1 * a_mat2; //the libraries required for this are not linking

	double diff = max_diff(mat3, mat4, M, N);
	std::cout << "diff between sequential and omp matrices: " << diff << std::endl;
	diff = max_diff(mat3, a_mat3.memptr(), M, N);
	std::cout << "diff between omp and armadillo results: " << diff << std::endl;


	print_matrix_big(mat3, M, N, 3);
	print_matrix_big(mat4, M, N, 3);
	print_matrix_big(a_mat3.memptr(), M, N, 3);
}