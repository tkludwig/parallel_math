/*goal: compare performance of a few sequential & parallel implementations of matrix multiplication*/

#include <hip/hip_runtime.h>
#include ""

#include <stdio.h>
#include <iostream>
#include <omp.h>
#include <chrono>
#include <random>
#include <iomanip>

#include "matrix_operations.cuh"

int main()
{
	const int M = 9;
	const int N = 9;
	double mat1[M * N];
	double mat2[M * N];
	double mat3[M * N];
	double mat4[M * N];
	random_uniform_initialize(mat1, M, N, 0.0, 0.5);
	random_uniform_initialize(mat2, M, N, 0.0, 0.5);

	std::cout << "mat1 initial:" << std::endl;
	print_matrix_big(mat1, M, N, 3);

	std::cout << "test omp naive multiply:" << std::endl;
	auto start = std::chrono::high_resolution_clock::now();
	naive_multiply_omp(mat1, mat2, mat3, M, N, N);
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_omp on size " << M << " took " << duration.count() << " ms" << std::endl;

	start = std::chrono::high_resolution_clock::now();
	naive_multiply_sequential(mat1, mat2, mat4, M, N, N);
	end = std::chrono::high_resolution_clock::now();
	duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
	std::cout << "naive_multiple_sequential on size " << M << " took " << duration.count() << " ms" << std::endl;

	print_matrix_big(mat3, M, N, 3);
	print_matrix_big(mat4, M, N, 3);
}